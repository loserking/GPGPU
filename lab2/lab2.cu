#include "lab2.h"
static const unsigned W = 640;
static const unsigned H = 480;
static const unsigned NFRAME = 240;

struct Lab2VideoGenerator::Impl {
	int t = 0;
};

Lab2VideoGenerator::Lab2VideoGenerator(): impl(new Impl) {
}

Lab2VideoGenerator::~Lab2VideoGenerator() {}

void Lab2VideoGenerator::get_info(Lab2VideoInfo &info) {
	info.w = W;
	info.h = H;
	info.n_frame = NFRAME;
	// fps = 24/1 = 24
	info.fps_n = 24;
	info.fps_d = 1;
};


void Lab2VideoGenerator::Generate(uint8_t *yuv) {
	hipMemset(yuv, (impl->t)*255/NFRAME, W*H);
	hipMemset(yuv+W*H, 128, W*H/2);
	++(impl->t);
}
